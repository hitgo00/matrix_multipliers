#include<stdio.h>
#include<time.h>
#include<math.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define SZ 16384
#define L 1024
#define T 32

__global__ void distribute(float *A11,float *A12,float *A21,float *A22,float *A,int K){

    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(y<K && x<K){
        A11[y*K + x] = A[y*K*2 + x]; //Because N = 2*K and A[y][x] = A[y*N + x] = A[y*K*2 + x] therefore we multiply with 2 to map correctly.
        A12[y*K + x] = A[y*K*2 + x + K];
        A21[y*K + x] = A[(y + K)*K*2 + x];
        A22[y*K + x] = A[(y + K)*K*2 + x + K];
    }
     __syncthreads();
 }
 __global__ void add(float *A, float *B, float *C, int K){
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(y<K && x<K){
        C[y*K + x] = A[y*K + x] + B[y*K + x];
    }
     __syncthreads();
 }
 __global__ void sub(float *A, float *B, float *C, int K){
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(y<K && x<K){
        C[y*K + x] = A[y*K + x] - B[y*K + x];
    }
     __syncthreads();
 }
 __global__ void gather(float *A11,float *A12,float *A21,float *A22,float *A,int K){
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(y<K && x<K){
        A[y*K*2 + x] = A11[y*K + x];
        A[y*K*2 + x + K] = A12[y*K + x];
        A[(y + K)*K*2 + x] = A21[y*K + x];
        A[(y + K)*K*2 + x + K] = A22[y*K + x];
    }
     __syncthreads();
 }
 __global__ void trivial_parallel(float *A, float* B, float* C, int N){
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(y<N && x<N){
        int i;
        float tot = 0;
        for(i=0;i<N;i++){
            tot+=A[y*N + i]*B[i*N + x];
        }
        C[y*N + x] = tot;
    }
     __syncthreads();
 }
void strassen(float *A, float *B, float *C,int N){

    float *A_G,*B_G,*C_G;

    int size_b = N*N*sizeof(float);
   

    hipMalloc((void**)&A_G, size_b);
    hipMalloc((void**)&B_G, size_b);
    hipMalloc((void**)&C_G, size_b);

    hipMemcpy(A_G,A,size_b,hipMemcpyHostToDevice);
    hipMemcpy(B_G,B,size_b,hipMemcpyHostToDevice);


    if(N<=L){
        dim3 dimblock(T,T);
        dim3 dimgrid((N+T-1)/T,(N+T-1)/T); 
        trivial_parallel<<<dimgrid,dimblock>>>(A_G,B_G,C_G,N);
    }
     
    else{
        int K = N/2; //Split into 4 parts so K = N/2 or N = 2*K

        int size_s = K*K*sizeof(float);

        float *A11,*A12,*A21,*A22,*B11,*B12,*B21,*B22,*C11,*C12,*C21,*C22,*T1,*T2; // Declare Device var
        
        hipMalloc((void**)&A11, size_s);
        hipMalloc((void**)&A12, size_s);
        hipMalloc((void**)&A21, size_s);
        hipMalloc((void**)&A22, size_s);

        hipMalloc((void**)&B11, size_s);
        hipMalloc((void**)&B12, size_s);
        hipMalloc((void**)&B21, size_s);
        hipMalloc((void**)&B22, size_s);

        hipMalloc((void**)&C11, size_s);
        hipMalloc((void**)&C12, size_s);
        hipMalloc((void**)&C21, size_s);
        hipMalloc((void**)&C22, size_s);

        hipMalloc((void **)&T1, size_s);
        hipMalloc((void **)&T2, size_s);

        
        dim3 dimblock(T,T);
        dim3 dimgrid((K+T-1)/T,(K+T-1)/T); 

        //Utilize only 2 temporary variables T1,T2 to reduce temporary storage per level significantly and avoid unnecessary data transfer.
        //Refer Sahni's paper.

        distribute<<<dimgrid,dimblock>>>(A11,A12,A21,A22,A_G,K);
        distribute<<<dimgrid,dimblock>>>(B11,B12,B21,B22,B_G,K);

        sub<<<dimgrid,dimblock>>>(A21,A11,C12,K);
        add<<<dimgrid,dimblock>>>(B11,B12,C21,K);
        strassen(C12,C21,C22,K);

        sub<<<dimgrid,dimblock>>>(A12,A22,C12,K);
        add<<<dimgrid,dimblock>>>(B21,B22,C21,K);
        strassen(C12,C21,C11,K);

        add<<<dimgrid,dimblock>>>(A11,A22,C12,K);
        add<<<dimgrid,dimblock>>>(B11,B22,C21,K);
        strassen(C12,C21,T1,K);

        add<<<dimgrid,dimblock>>>(T1,C11,C11,K);
        add<<<dimgrid,dimblock>>>(T1,C22,C22,K);
        add<<<dimgrid,dimblock>>>(A21,A22,T2,K);
        strassen(T2,B11,C21,K);

        sub<<<dimgrid,dimblock>>>(C22,C21,C22,K);
        sub<<<dimgrid,dimblock>>>(B21,B11,T1,K);
        strassen(A22,T1,T2,K);

        add<<<dimgrid,dimblock>>>(C21,T2,C21,K);
        add<<<dimgrid,dimblock>>>(C11,T2,C11,K);
        sub<<<dimgrid,dimblock>>>(B12,B22,T1,K);
        strassen(A11,T1,C12,K);

        add<<<dimgrid,dimblock>>>(C22,C12,C22,K);
        add<<<dimgrid,dimblock>>>(A11,A12,T2,K);
        strassen(T2,B22,T1,K);

        add<<<dimgrid,dimblock>>>(C12,T1,C12,K);
        sub<<<dimgrid,dimblock>>>(C11,T1,C11,K);

        gather<<<dimgrid,dimblock>>>(C11,C12,C21,C22,C_G,K);
        

        hipFree(A11); 
        hipFree(A12); 
        hipFree(A21); 
        hipFree(A22); 
        hipFree(B11); 
        hipFree(B12); 
        hipFree(B21); 
        hipFree(B22); 
        hipFree(T1);
        hipFree(T2);	
    }
    hipMemcpy(C, C_G, size_b, hipMemcpyDeviceToHost);
    hipFree(A_G);
	hipFree(B_G);
	hipFree(C_G);

}
int main(){
    int size = SZ*SZ*sizeof(float);

    float *A,*B,*C,*check;

    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);
    check = (float*)malloc(size);

    int i,j,k;

    for(i=0;i<SZ;i++){
        for(j=0;j<SZ;j++){
            A[i * SZ + j] = 20*((float)rand() / (float)RAND_MAX);
			B[i * SZ + j] = 20*((float)rand() / (float)RAND_MAX); //Random Floating values between 0...20
			C[i * SZ + j] = 0;
			check[i * SZ + j] = 0;
        }
    }
    /*
    for(i = 0; i < SZ; i++) {
		for(j = 0; j < SZ; j++) {
			for(k = 0; k < SZ; k++) {
				check[i * SZ + j] += A[i * SZ + k] * B[k * SZ + j];
			}
		}
    }
    */

    //TODO : Make clock more accurate.
    clock_t t_f;

    t_f = clock();
         strassen(A, B, C, SZ);
    t_f = clock() - t_f;


    /*
    for(i=0;i<SZ*SZ;++i){
        printf("%f %f \n",C[i],check[i]);
    }
    */
    
   // printf("--------------------------------------------------------------------------------------------------------------------------------------------\n");

    double time_taken = ((double)t_f)/CLOCKS_PER_SEC;

    printf("time(s) = %lf \n", time_taken);

    /*
        N       Leaf     Time(s)
        <1024     -       0.00001
        1024     128      1.25000
        2048     128      2.25000
        8192     128      98.77000
        8192     512      15.99000
        8192     1024      8.74000
        16384    1024     16.43000
        16384    2048     30.93000
        16384    4096     54.79000
    */
}